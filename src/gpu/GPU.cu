#include "hip/hip_runtime.h"
#include <algorithm>
#include <stdint.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/binary_search.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <mutex>
#include <vector>
#include <sys/time.h>
#include <condition_variable>
#include "GPU.h"

#include "../stdexcept.hpp"


#define BAND_SIZE 32 
#define LOG_BLOCK_SIZE 7

#define BLOCK_SIZE (1 << LOG_BLOCK_SIZE)

int NUM_BLOCKS;
size_t BATCH_SIZE;

std::mutex mu;
std::condition_variable cv;
std::vector<int> available_gpus;

uint32_t num_unique_markers;

uint32_t** d_alignments;
uint32_t** d_score;
uint32_t** d_score_pos;
uint32_t** d_num_traceback;
uint32_t** d_common_markers;
uint32_t** d_num_common_markers;
uint64_t** d_batch_rid_markers;

using namespace shasta;

__global__
void initialize_batch_rid_markers (uint64_t* batch_rid_markers, uint32_t num_unique_markers, uint32_t batch_size) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    for (uint64_t i=bx; i < 2*batch_size; i+=gs) {
        uint64_t v, val;
        v = (i << (32+SHASTA_LOG_MAX_MARKERS_PER_READ));
        for (uint64_t j=tx; j< num_unique_markers; j+=bs) {
            val = v + (j << SHASTA_LOG_MAX_MARKERS_PER_READ);
            batch_rid_markers[i*num_unique_markers+j] = val;
        }
    }
    if (bx==0) {
        if (tx == 0) {
            uint64_t v = 2*batch_size;
            batch_rid_markers[2*batch_size*num_unique_markers] = (v << (32+SHASTA_LOG_MAX_MARKERS_PER_READ));
        }
    }
}

__global__
void find_common_markers (uint64_t maxMarkerFrequency, uint64_t n, uint32_t num_unique_markers, uint64_t* read_pairs, uint64_t* index_table, uint64_t* rid_marker_pos, uint64_t* sorted_rid_marker_pos, uint32_t* num_common_markers, uint32_t* common_markers)
{
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    uint64_t m_mask = ((uint64_t) 1 << 32) - 1;
    uint64_t p_mask = ((uint64_t) 1 << SHASTA_LOG_MAX_MARKERS_PER_READ) - 1;
    
    __shared__ uint32_t prefix[1+BLOCK_SIZE];

    __syncthreads();

    for (int i = bx; i < n; i+=gs) {
        if (tx == 0) {
            prefix[tx] = i*SHASTA_MAX_COMMON_MARKERS_PER_READ;
            num_common_markers[i] = 0;
        }
        __syncthreads();

        uint64_t v1 = read_pairs[2*i];
        uint64_t v2 = read_pairs[2*i+1];
        uint64_t rid1 = (v1 >> 32);
        uint64_t rid2 = (v2 >> 32);
        uint64_t l1 = ((v1 << 32) >> 32);
        uint64_t l2 = ((v2 << 32) >> 32);

        if ((l1 > 0) && (l2 > 0)) {
            uint64_t s2 = index_table[rid2*num_unique_markers];
            uint64_t e2 = s2+l2;

            for (uint64_t j = s2; j < e2; j += bs) {
                uint64_t idx = tx+j;
                uint64_t marker;
                uint64_t sm1=0, sm2=0, em1=0, em2=0;

                prefix[1+tx] = 0; 

                if (idx < e2) {
                    uint64_t v = rid_marker_pos[idx];
                    marker = ((v >> SHASTA_LOG_MAX_MARKERS_PER_READ) & m_mask);

                    sm1 = index_table[rid1*num_unique_markers+marker];
                    em1 = index_table[rid1*num_unique_markers+marker+1];
                    sm2 = index_table[rid2*num_unique_markers+marker];
                    em2 = index_table[rid2*num_unique_markers+marker+1];

                    if ((em1 - sm1 <= maxMarkerFrequency) && (em2 - sm2 <= maxMarkerFrequency)) {
                        prefix[1+tx] = (em1-sm1);
                    }
                }

                __syncthreads();

                if (tx == 0) {
                    for (int r = 0; r < BLOCK_SIZE; r++) {
                        prefix[1+r] += prefix[r];
                    }
                }

                __syncthreads();

                uint32_t mhs = prefix[tx];
                uint32_t mhe = prefix[1+tx];

                for (uint64_t k1 = 0; k1 < (mhe-mhs); k1++) {
                    if (mhs+k1 < (i+1)*SHASTA_MAX_COMMON_MARKERS_PER_READ) {
                        uint64_t sv1 = sorted_rid_marker_pos[sm1+k1];
                        uint32_t cm = (sv1 & p_mask) + 1;
                        cm = (cm << 16) + (1+idx-s2);
                        common_markers[mhs+k1] = cm;
                    }
                }

                __syncthreads();

                if (tx == 0) {
                    prefix[tx] = prefix[BLOCK_SIZE];
                }

                __syncthreads();
            }

            if (tx == 0) {
                uint32_t num_common = prefix[tx] - i*SHASTA_MAX_COMMON_MARKERS_PER_READ;
                num_common_markers[i] = num_common;
            }
        }

        __syncthreads();
    }
}

__global__
void find_traceback (int n, size_t maxSkip, uint32_t* d_score, uint32_t* d_common_markers, uint32_t* d_num_common_markers, uint32_t* d_score_pos, uint32_t* d_alignments, uint32_t* d_num_traceback) {
    int tx = threadIdx.x;
    int bs = blockDim.x;
    int bx = blockIdx.x;
    int gs = gridDim.x;

    __shared__ uint32_t score[BAND_SIZE];
    __shared__ uint32_t score_pos[BAND_SIZE];
    __shared__ int num_common_markers;
    __shared__ bool stop_shared;

    for (int i = bx; i < n; i += gs) {
        uint32_t max_score = 0, max_score_pos = 0;
        uint32_t addr1 = i*SHASTA_MAX_COMMON_MARKERS_PER_READ;
        uint32_t addr2 = bx*SHASTA_MAX_COMMON_MARKERS_PER_READ;
        uint32_t addr3 = i*SHASTA_MAX_TB;

        if (tx == 0) {
            num_common_markers = d_num_common_markers[i];
            if (num_common_markers >= SHASTA_MAX_COMMON_MARKERS_PER_READ) {
                num_common_markers = 0;
            }
            d_alignments[addr3] = 0;
        }
        score[tx] = 0;

        __syncthreads();

        for (int p = 0; p < num_common_markers; p++) {
            uint32_t v = d_common_markers[addr1+p];
            uint32_t l = ((v << 16) >> 16);
            uint32_t u = (v >> 16);

            int ptr = p - tx - 1;

            score[tx] = 1;
            score_pos[tx] = p;

            bool stop = false;
            __syncthreads();

            while (!stop) {
                uint32_t l1, u1;
                if (ptr >= 0) {
                    uint32_t v1 = d_common_markers[addr1+ptr];
                    l1 = ((v1 << 16) >> 16);
                    u1 = (v1 >> 16);
                    if ((l1 < l) && (u1 < u) && (u-u1 <= maxSkip) && (l-l1 <= maxSkip)) {
                        uint32_t pscore = d_score[addr2+ptr];
                        if (score[tx] < pscore+1) { 
                            score[tx] = pscore+1;
                            score_pos[tx] = ptr;
                        }
                    }
                }
                ptr -= bs;
                if (tx == bs-1) {
                    if ((ptr < 0) || (l-l1 > maxSkip))  {
                        stop_shared = true;
                    }
                    else {
                        stop_shared = false;
                    }
                }
                __syncthreads();
                stop = stop_shared;
            }

            __syncthreads();

            // parallel reduction (max)
            for(unsigned int s = 1; s < bs; s *= 2) {
                if (tx % (2*s) == 0) {
                    if (score[tx] < score[tx+s]) { 
                        score[tx] = score[tx + s];
                        score_pos[tx] = score_pos[tx + s];
                    }
                }
                __syncthreads();
            }
            
            if (tx == 0) {
                d_score[addr2+p] = score[0];
                d_score_pos[addr2+p] = score_pos[0];
                if (score[0] > max_score) {
                    max_score = score[0];
                    max_score_pos = score_pos[0];
                }
            }
            __syncthreads();
        }

        __syncthreads();

        if (tx == 0) {
            int num_ptr = 0;

            if (max_score > 0) {
                int curr_pos = max_score_pos;
                int prev_pos = max_score_pos + 1;

                while ((curr_pos >= 0) && (prev_pos > curr_pos)) {
                    prev_pos = curr_pos;
                    if (num_ptr < SHASTA_MAX_TB) {
                        d_alignments[addr3+num_ptr] = d_common_markers[addr1+curr_pos];
                    }
                    num_ptr++;
                    curr_pos = d_score_pos[addr2+curr_pos];
                }
            }

            if (num_ptr < SHASTA_MAX_TB) {
                d_alignments[addr3+num_ptr] = 0;
                d_num_traceback[i] = num_ptr;
            }
            else {
                d_alignments[addr3] = 0;
                d_num_traceback[i] = 0;
            }
        }
        __syncthreads();
    }
}

extern "C" std::tuple<int, size_t> shasta_initializeProcessors (size_t numUniqueMarkers) {
    int nDevices;

    num_unique_markers = (uint32_t) numUniqueMarkers;

    hipGetDeviceCount(&nDevices);
    
    size_t device_memory;
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        device_memory = prop.totalGlobalMem;
        if (device_memory > 0xffffffff) {
            NUM_BLOCKS = (1 << 10);
            BATCH_SIZE = (1 << 11);
        }
        else {
            NUM_BLOCKS = (1 << 8);
            BATCH_SIZE = (1 << 9);
            break;
        }
        //printf("Device Number: %d\n", i);
        //printf("  Device name: %s\n", prop.name);
        //printf("  Memory Clock Rate (KHz): %d\n",
        //prop.memoryClockRate);
        //printf("  Memory Bus Width (bits): %d\n",
        //prop.memoryBusWidth);
        //printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
        //2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    d_alignments = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));

    d_score = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_score_pos = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_traceback = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_num_common_markers = (uint32_t**) malloc(nDevices*sizeof(uint32_t*));
    d_batch_rid_markers = (uint64_t**) malloc(nDevices*sizeof(uint64_t*));

    hipError_t err;
    size_t num_bytes;

    for (int k=0; k<nDevices; k++) {
        
        available_gpus.push_back(k);

        err = hipSetDevice(k);
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: could not set device");
        }
        
        num_bytes = BATCH_SIZE*SHASTA_MAX_TB*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_alignments[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        num_bytes = NUM_BLOCKS*SHASTA_MAX_COMMON_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_score[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = NUM_BLOCKS*SHASTA_MAX_COMMON_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_score_pos[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_traceback[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = BATCH_SIZE*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_num_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }
        
        num_bytes = BATCH_SIZE*SHASTA_MAX_COMMON_MARKERS_PER_READ*sizeof(uint32_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_common_markers[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        num_bytes = (1+2*BATCH_SIZE*numUniqueMarkers)*sizeof(uint64_t);
        if (k==0)
            fprintf(stdout, "\t-Requesting %3.0e bytes on GPU\n", (double)num_bytes);
        err = hipMalloc(&d_batch_rid_markers[k], num_bytes); 
        if (err != hipSuccess) {
            throw runtime_error("GPU_ERROR: hipMalloc failed!\n");
        }

        initialize_batch_rid_markers<<<NUM_BLOCKS, BLOCK_SIZE>>> (d_batch_rid_markers[k], numUniqueMarkers, BATCH_SIZE);  
    }

    return std::make_tuple(nDevices, BATCH_SIZE);
}

extern "C" void shasta_alignBatchGPU (size_t maxMarkerFrequency, size_t maxSkip, size_t n, uint64_t num_pos, uint64_t num_reads, uint64_t* batch_rid_marker_pos, uint64_t* batch_read_pairs, uint32_t* h_alignments, uint32_t* h_num_traceback) {
    bool report_time = false;

    int k = -1;

    while (k < 0) {
        std::unique_lock<std::mutex> locker(mu);
        if (available_gpus.empty()) {
            cv.wait(locker, [](){return !available_gpus.empty();});
        }
        k = available_gpus.back();
        available_gpus.pop_back();
        locker.unlock();
    }

    struct timeval t1, t2, t3;
    long useconds, seconds, mseconds;
    
    hipError_t err; 

    err = hipSetDevice(k);
    if (err != hipSuccess) {
        throw runtime_error("GPU_ERROR: could not set device.\n");
    }
    
    gettimeofday(&t1, NULL);

    try {
        thrust::device_ptr<uint64_t> d_batch_rid_markers_ptr = thrust::device_pointer_cast(d_batch_rid_markers[k]);

        thrust::device_vector<uint64_t> t_d_rid_marker_pos (batch_rid_marker_pos, batch_rid_marker_pos + num_pos);
        thrust::device_vector<uint64_t> t_d_sorted_rid_marker_pos (batch_rid_marker_pos, batch_rid_marker_pos+num_pos);
        thrust::device_vector<uint64_t> t_d_rid_markers (d_batch_rid_markers_ptr, d_batch_rid_markers_ptr+num_reads*num_unique_markers+1);
        thrust::device_vector<uint64_t> t_d_read_pairs (batch_read_pairs, batch_read_pairs+2*n);
        thrust::device_vector<uint64_t> t_d_index_table (num_reads*num_unique_markers+1);

        thrust::sort(t_d_sorted_rid_marker_pos.begin(), t_d_sorted_rid_marker_pos.end());


        thrust::lower_bound(t_d_sorted_rid_marker_pos.begin(),
                t_d_sorted_rid_marker_pos.end(),
                t_d_rid_markers.begin(),
                t_d_rid_markers.end(),
                t_d_index_table.begin());

        gettimeofday(&t2, NULL);

        uint64_t* d_sorted_rid_marker_pos = thrust::raw_pointer_cast (t_d_sorted_rid_marker_pos.data());
        uint64_t* d_rid_marker_pos = thrust::raw_pointer_cast (t_d_rid_marker_pos.data());
        uint64_t* d_index_table = thrust::raw_pointer_cast (t_d_index_table.data());
        uint64_t* d_read_pairs = thrust::raw_pointer_cast (t_d_read_pairs.data());

        find_common_markers <<<NUM_BLOCKS, BLOCK_SIZE>>> (maxMarkerFrequency, n, num_unique_markers, d_read_pairs, d_index_table, d_rid_marker_pos, d_sorted_rid_marker_pos, d_num_common_markers[k], d_common_markers[k]);
        
        find_traceback <<<NUM_BLOCKS, BAND_SIZE>>>(n, maxSkip, d_score[k], d_common_markers[k], d_num_common_markers[k], d_score_pos[k], d_alignments[k], d_num_traceback[k]);

    }
    catch (std::bad_alloc) {
        throw runtime_error("Insufficient GPU memory. Try on GPU with larger memory or without --gpu option.\n");
    }

    err = hipMemcpy(h_num_traceback, d_num_traceback[k], n*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw runtime_error("Error: hipMemcpy failed!\n");
    }

    err = hipMemcpy(h_alignments, d_alignments[k], n*SHASTA_MAX_TB*sizeof(uint32_t), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw runtime_error("Error: hipMemcpy failed!\n");
    }
    
    {
        std::unique_lock<std::mutex> locker(mu);
        available_gpus.push_back(k);
        locker.unlock();
        cv.notify_one();
    }
    
    gettimeofday(&t3, NULL);
    
    if (report_time) {
        useconds = t2.tv_usec - t1.tv_usec;
        seconds = t2.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;
        fprintf(stderr, "Time elapsed (t2-t1): %ld msec \n", mseconds);

        useconds = t3.tv_usec - t1.tv_usec;
        seconds = t3.tv_sec - t1.tv_sec;
        mseconds = ((seconds) * 1000 + useconds/1000.0) + 0.5;
        fprintf(stderr, "Time elapsed (t3-t1): %ld msec \n", mseconds);
    }

    return;
}

extern "C" void shasta_shutdownProcessors(int nDevices) {
    hipDeviceReset();
}
